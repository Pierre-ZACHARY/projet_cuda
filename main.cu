#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <opencv2/opencv.hpp>
using namespace cv;

__global__ void vecadd( int * v1, int * v2 )
{
    auto tid = threadIdx.x;

    v2[tid] += v1[tid];

}


int main()
{

    int count = 0;

    hipGetDeviceCount( &count );

    std::cout << count << " device(s) found.\n";

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::cout << "maxBlocksPerMultiProcessor " << prop.maxBlocksPerMultiProcessor << std::endl;
    std::cout << "maxThreadsPerBlock " << prop.maxThreadsPerBlock << std::endl;
    std::cout << "multiProcessorCount " << prop.multiProcessorCount << std::endl;
    std::cout << "warpSize " << prop.warpSize << std::endl;

    Mat m_in = imread("../in.jpg", IMREAD_UNCHANGED );
    std::cout << "Image Size : " << m_in.rows * m_in.cols << std::endl;


    std::vector< int > v1( 10 );
    std::vector< int > v2( 10 );

    int * v1_d = nullptr;
    int * v2_d = nullptr;

    for( std::size_t i = 0 ; i < v1.size() ; ++i )
    {
        v1[ i ] =  5;
        v2[ i ] =  4;
    }

    hipMalloc( &v1_d, v1.size() * sizeof( int ) );
    hipMalloc( &v2_d, v2.size() * sizeof( int ) );

    hipMemcpy(v1_d, v1.data(), v1.size() * sizeof( int ), hipMemcpyHostToDevice );
    hipMemcpy(v2_d, v2.data(), v2.size() * sizeof( int ), hipMemcpyHostToDevice );

    vecadd<<< 1, 10 >>>( v1_d, v2_d );

    hipMemcpy(v2.data(),v2_d, v2.size() * sizeof( int ), hipMemcpyDeviceToHost );



    for (size_t idex = 0; idex < v1.size(); idex++)
        std::cout <<   v2[idex] << " ";
    std::cout << std::endl;



    hipFree( v1_d );
    hipFree( v2_d );

    return 0;
}
